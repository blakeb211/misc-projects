#include <iostream>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition
__global__ void vectorAdd(const float* a, const float* b, float* c, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        c[idx] = a[idx] + b[idx];
    }
}

bool verify_device_avail() {
    int deviceCount{0};
    hipGetDeviceCount(&deviceCount);
    std::cout << "Number of working devices found: " << deviceCount << "\n";
    if (deviceCount > 0) {
        return true;
    } else {
        return false;
    }
}

int main() {
    if (verify_device_avail()) {
        hipSetDevice(0);
    }
    const int size = 1024; // Size of the arrays
    const int threadsPerBlock = 256;
    const int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    float* hostA = new float[size];
    float* hostB = new float[size];
    float* hostC = new float[size];

    // Initialize arrays
    for (int i = 0; i < size; i++) {
        hostA[i] = static_cast<float>(i);
        hostB[i] = static_cast<float>(2 * i);
    }

    // Device (GPU) arrays
    float* deviceA;
    float* deviceB;
    float* deviceC;

    // Allocate memory on the GPU
    hipMalloc((void**)&deviceA, size * sizeof(float));
    hipMalloc((void**)&deviceB, size * sizeof(float));
    hipMalloc((void**)&deviceC, size * sizeof(float));

    // Copy data from the host to the device
    hipMemcpy(deviceA, hostA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, size * sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel on the GPU
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(deviceA, deviceB, deviceC, size);

    // Copy the result from the device to the host
    hipMemcpy(hostC, deviceC, size * sizeof(float), hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < size; i++) {
        std::cout << hostA[i] << " + " << hostB[i] << " = " << hostC[i] << std::endl;
    }

    // Free device and host memory
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
    delete[] hostA;
    delete[] hostB;
    delete[] hostC;

    return 0;
}

